//============================================================================
// Name        : parallelization1.cpp
// Author      : 
// Version     :
// Copyright   : Your copyright notice
// Description : Hello World in C++, Ansi-style
//============================================================================


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
  /*
  for(int i = 0; i < N; ++i)
  {
    result[i] = a[i] + b[i];
  }
  */

	size_t stride_s = gridDim.x * blockDim.x;

	for(int i = threadIdx.x + blockDim.x*blockIdx.x; i<N; i = i + stride_s)
	{
		result[i] = a[i] + b[i];
	}


}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}


hipError_t cudaRunCheck(hipError_t result)
{
	if(result != hipSuccess)
	{
		printf("Error: %s\n", hipGetErrorString(result));
		assert(result == hipSuccess);
	}

	return result;

}


int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;
/*
  a = (float *)malloc(size);
  b = (float *)malloc(size);
  c = (float *)malloc(size);
*/

  cudaRunCheck(hipMallocManaged(&a, size));
  cudaRunCheck(hipMallocManaged(&b, size));
  cudaRunCheck(hipMallocManaged(&c, size));

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  size_t nbrThreadsBlock = 1024;
  size_t nbrBlocks = (N + nbrThreadsBlock -1)/nbrThreadsBlock;

  addVectorsInto<<<nbrBlocks, nbrThreadsBlock>>>(c, a, b, N);

  hipDeviceSynchronize();

  hipError_t kernelRun_err = hipGetLastError();

  if(kernelRun_err != hipSuccess)
  {
	  printf("Error kernel launch: %s", hipGetErrorString(kernelRun_err));
  }


  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
