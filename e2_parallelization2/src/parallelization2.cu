//============================================================================
// Name        : parallelization1.cpp
// Author      : 
// Version     :
// Copyright   : Your copyright notice
// Description : Hello World in C++, Ansi-style
//============================================================================


#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */

__global__ void loop(int N)
{
/*
  for (int i = 0; i < N; ++i)
  {
    printf("This is iteration number %d\n", i);
  }
  */

  printf("This is iteration number %d\n", blockIdx.x*blockDim.x + threadIdx.x);

}

int main()
{
  /*
   * When refactoring `loop` to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * For this exercise, be sure to use more than 1 block in
   * the execution configuration.
   */

  int N = 10;
  loop<<<2, 5>>>(N);

  hipDeviceSynchronize();
}
